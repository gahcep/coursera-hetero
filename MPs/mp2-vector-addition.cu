#include "hip/hip_runtime.h"
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < len)
		out[idx] = in1[idx] + in2[idx];
}

int main(int argc, char **argv) {
	wbArg_t args;
	int inputLength;
	float *hostInput1;
	float *hostInput2;
	float *hostOutput;
	float *deviceInput1;
	float *deviceInput2;
	float *deviceOutput;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
	hostOutput = (float *)malloc(inputLength * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);

	int size = inputLength * sizeof(float);

	wbTime_start(GPU, "Allocating GPU memory.");
	hipMalloc((void **)&deviceInput1, size);
	hipMalloc((void **)&deviceInput2, size);
	hipMalloc((void **)&deviceOutput, size);
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	dim3 DimGrid((inputLength - 1) / 256 + 1, 1, 1);
	dim3 DimBlock(256, 1, 1);

	wbTime_start(Compute, "Performing CUDA computation");
	vecAdd << <DimGrid, DimBlock >> >(deviceInput1, deviceInput2, deviceOutput, inputLength);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, inputLength);

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

	return 0;
}
