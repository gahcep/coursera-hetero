#include	<wb.h>

int main(int argc, char ** argv) {
	int deviceCount;

	wbArg_read(argc, argv);

	hipGetDeviceCount(&deviceCount);

	wbTime_start(GPU, "Getting GPU Data."); //@@ start a timer

	for (int dev = 0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;

		hipGetDeviceProperties(&deviceProp, dev);

		if (dev == 0) {
			if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
				wbLog(TRACE, "No CUDA GPU has been detected");
				return -1;
			}
			else if (deviceCount == 1) {
				wbLog(TRACE, "There is 1 device supporting CUDA");
			}
			else {
				wbLog(TRACE, "There are ", deviceCount, " devices supporting CUDA");
			}
		}

		wbLog(TRACE, "Device ", dev, " name: ", deviceProp.name);
		wbLog(TRACE, " Computational Capabilities: ", deviceProp.major, ".", deviceProp.minor);
		wbLog(TRACE, " Maximum global memory size: ", deviceProp.totalGlobalMem);
		wbLog(TRACE, " Maximum constant memory size: ", deviceProp.totalConstMem);
		wbLog(TRACE, " Maximum shared memory size per block: ", deviceProp.sharedMemPerBlock);
		wbLog(TRACE, " Maximum block dimensions: ", deviceProp.maxThreadsDim[0], " x ",
			deviceProp.maxThreadsDim[1], " x ",
			deviceProp.maxThreadsDim[2]);
		wbLog(TRACE, " Maximum grid dimensions: ", deviceProp.maxGridSize[0], " x ",
			deviceProp.maxGridSize[1], " x ",
			deviceProp.maxGridSize[2]);
		wbLog(TRACE, " Warp size: ", deviceProp.warpSize);
	}

	wbTime_stop(GPU, "Getting GPU Data.");

	return 0;
}

