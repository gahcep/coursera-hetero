#include "hip/hip_runtime.h"


int main()
{
	hipError_t err;

	// Device number
	int deviceCount = 0;
	err = hipGetDeviceCount(&deviceCount);

	if (err != hipSuccess)
		return 1;

	/*
		CUDA 3.0
		totalGlobalMem = 2GB
		sharedMemPerBlock = 49152 bytes
		regsPerBlock = 65536
		warpSize = 32
		maxThreadsPerBlock = 1024	
		maxThreadsDim = {1024, 1024, 64}
		maxGridSize	= {2147483647, 65535, 65535}
		totalConstMem = 65536
	*/

	hipDeviceProp_t props;
	for (int i = 0; i < deviceCount; i++)
	{
		err = hipGetDeviceProperties(&props, i);
		if (err != hipSuccess)
			return 1;
	}
}
